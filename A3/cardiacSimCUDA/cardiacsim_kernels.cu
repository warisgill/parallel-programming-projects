/* 
 * Solves the Panfilov model using an explicit numerical scheme.
 * Based on code orginally provided by Xing Cai, Simula Research Laboratory 
 * and reimplementation by Scott B. Baden, UCSD
 * 
 * Modified and  restructured by Didem Unat, Koc University
 *
 * Refer to "Detailed Numerical Analyses of the Aliev-Panfilov Model on GPGPU"
 * https://www.simula.no/publications/detailed-numerical-analyses-aliev-panfilov-model-gpgpu
 * by Xing Cai, Didem Unat and Scott Baden
 *
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <stdlib.h>
#include <iostream>
#include <iomanip>
#include <string.h>
#include <math.h>
#include <sys/time.h>
#include <getopt.h>
using namespace std;

#define TILE_DIM 32
#define kk  8.0  
#define a 0.1 
#define epsilon 0.01 
#define M1 0.07 
#define M2 0.3 
#define b  0.1
#define d  5e-5



// For Command Line Args
void cmdLine(int argc, char *argv[], double &T, int &n, int &px, int &py, int &plot_freq, int &no_comm, int &num_threads);
// Timer: Make successive calls and take a difference to get the elapsed time.
double getTime();
// Allocate a 2D array
double **alloc2D(int m, int n);
// Mirror Ghost Boundries
void mirrorBoundries(double *E_prev_1D, const int n, const int m, const int WIDTH);
void mirrorBoundries(double *E_prev_1D, double* d_E_prev_1D, const int n, const int m, const int WIDTH);
/* 
	Reports statistics about the computation
	These values should not vary (except to within roundoff)
 	when we use different numbers of  processes to solve the problem
*/

double stats(double **E, int m, int n, double *_mx);
double stats1D(double *E, int m, int n, double *_mx, int WIDTH);


// ============================== Kernels  ===========================
__global__ void mirrorkernel(double *E_prev_1D, const int n, const int m, const int WIDTH);
void simV1(const double alpha, const int n, const int m,  const double dt,  int WIDTH, double* time, double *d_E_1D, double *d_E_prev_1D, double *d_R_1D);
void simV2(const double alpha, const int n, const int m,  const double dt,  int WIDTH, double* time, double *d_E_1D, double *d_E_prev_1D, double *d_R_1D);
void simV3(const double alpha, const int n, const int m,  const double dt,  int WIDTH, double* time, double *d_E_1D, double *d_E_prev_1D, double *d_R_1D);
void simV4(const double alpha, const int n, const int m,  const double dt,  int WIDTH, double* time, double *d_E_1D, double *d_E_prev_1D, double *d_R_1D);
void simV5(const double alpha, const int n, const int m,  const double dt,  int WIDTH, double* time, double *d_E_1D, double *d_E_prev_1D, double *d_R_1D);
// ============================= Exp 1= ===============



// Main Refined -- Versioin 4 Refined --

// Main program
int main(int argc, char **argv)
{
	/*
	*  Solution arrays
	*   E is the "Excitation" variable, a voltage
	*   R is the "Recovery" variable
	*   E_prev is the Excitation variable for the previous timestep,
	*      and is used in time integration
	*/

	// For Serial Version
	double **E, **R, **E_prev;
	// For Host and GPU 
	double *E_1D, *R_1D, *E_prev_1D;
	double *d_E_1D, *d_E_prev_1D, *d_R_1D;

	// Various constants - these definitions shouldn't change

	double T = 1000.0;
	int m = 200, n = 200;
	int plot_freq = 0;
	int px = 1, py = 1;
	int no_comm = 0;
	int version = 4;
	int WIDTH;
	double time_elapsed = 0.0;
	
	// int version = 4;

	cmdLine(argc, argv, T, n, px, py, plot_freq, no_comm, version);
	m = n;
	
	// Allocate contiguous memory for solution arrays
	// The computational box is defined on [1:m+1,1:n+1]
	// We pad the arrays in order to facilitate differencing on the
	// boundaries of the computation box
	int Total_Bytes = (m + 2) * (n + 2) * sizeof(double);
	WIDTH = m + 2;

	E = alloc2D(m + 2, n + 2);
	E_prev = alloc2D(m + 2, n + 2);
	R = alloc2D(m + 2, n + 2);

	// Allocate space on the host (PINNED Memory)
	hipError_t status = hipHostMalloc(&E_1D, Total_Bytes, hipHostMallocDefault);
	status = hipHostMalloc(&E_prev_1D, Total_Bytes, hipHostMallocDefault);
	status = hipHostMalloc(&R_1D, Total_Bytes, hipHostMallocDefault);

	if (status != hipSuccess) {
		printf("Error allocating pinned host memory\n");
	}
			
	// Allocate space on the GPU
	hipMalloc(&d_E_1D, Total_Bytes);
	hipMalloc(&d_E_prev_1D, Total_Bytes);
	hipMalloc(&d_R_1D, Total_Bytes);

	
	int col, row;
	// Initialization
	for (row = 1; row <= m; row++)
	{
		for (col = 1; col <= n; col++)
		{
			E_prev[row][col] = 0;
			R[row][col] = 0;
			E_prev_1D[row * WIDTH + col] = 0;
			R_1D[row * WIDTH + col] = 0;
		}
	}

	for (row = 1; row <= m; row++)
	{
		for (col = n / 2 + 1; col <= n; col++)
		{
			E_prev[row][col] = 1.0;
			E_prev_1D[row * WIDTH + col] = 1.0;
		}
	}

	for (row = m / 2 + 1; row <= m; row++)
	{
		for (col = 1; col <= n; col++)
		{
			R[row][col] = 1.0;
			R_1D[row * WIDTH + col] = 1.0;
		}
	}

	
	double dx = 1.0 / n;

	// For time integration, these values shouldn't change
	double rp = kk * (b + 1) * (b + 1) / 4;
	double dte = (dx * dx) / (d * 4 + ((dx * dx)) * (rp + kk));
	double dtr = 1 / (epsilon + ((M1 / M2) * rp));
	double dt = (dte < dtr) ? 0.95 * dte : 0.95 * dtr;
	double alpha = d * dt / (dx * dx);
	int devId = 0;
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, devId);


	printf("\n    ******** Device : %s **********\n", prop.name);
	cout << "Simulation Version          : " << version<<endl;
	cout << "Block Size      :"<< TILE_DIM <<endl;
	cout << "Grid Size       : " << n << endl;
	cout << "Duration of Sim : " << T << endl;
	cout << "Time step dt    : " << dt << endl;
	cout << "Process geometry: " << px << " x " << py << endl;

	if (no_comm)
	{
		cout << "Communication   : DISABLED" << endl;
	}
	cout << endl;

	// Start the timer
	//double t0 = getTime();

	// Simulated time is different from the integer timestep number
	// Simulated time
	double t = 0.0;
	// Integer timestep number
	int niter = 0;
	const dim3 block_size(TILE_DIM, TILE_DIM);
	const dim3 num_blocks(WIDTH / block_size.x, WIDTH / block_size.y);
	
	hipMemcpy(d_R_1D, R_1D, Total_Bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_E_prev_1D, E_prev_1D, Total_Bytes, hipMemcpyHostToDevice);
	// very well done
	//simV5(alpha, n, m,  dt, WIDTH, &time_elapsed, d_E_1D, d_E_prev_1D, d_R_1D);

	while (t < T)
	{

		t += dt;
		niter++;

		//mirrorBoundries(E_prev_1D, n, m, WIDTH);
		//mirrorBoundries(E_prev_1D, d_E_prev_1D,n, m, WIDTH);
		

		mirrorkernel<<<num_blocks, block_size>>>(d_E_prev_1D, n, m ,WIDTH);
		hipStreamSynchronize(0);
		//cudaMemcpy(E_prev_1D, d_E_prev_1D, Total_Bytes, cudaMemcpyDeviceToHost);
		switch (version){
			case 1:
				simV1(alpha, n, m,  dt, WIDTH, &time_elapsed, d_E_1D, d_E_prev_1D, d_R_1D);
				break;
			case 2:
				simV2(alpha, n, m,  dt, WIDTH, &time_elapsed, d_E_1D, d_E_prev_1D, d_R_1D);
				break;
			case 3:
				simV3(alpha, n, m,  dt, WIDTH, &time_elapsed, d_E_1D, d_E_prev_1D, d_R_1D);
				break;
			case 4:
				
				simV4(alpha, n, m,  dt, WIDTH, &time_elapsed, d_E_1D, d_E_prev_1D, d_R_1D);
				break;
			// case 5:	
				
			// 	break;
			case 0:

			cout<<"\n Implement the Serial Version"<<endl;		
				break;
			default:
				cout<<"\nPlease Enter the Correct version"<<endl;
				return 0;
				
		}
		
		//cudaMemcpy(d_E_prev_1D, d_E_1D, Total_Bytes, cudaMemcpyDeviceToDevice);

		//swap current E with previous E
		double **tmp = E;
		E = E_prev;
		E_prev = tmp;

		double *tmp2 = d_E_1D;
		d_E_1D = d_E_prev_1D;
		d_E_prev_1D = tmp2;

	} //end of while loop

	hipMemcpy(E_prev_1D, d_E_prev_1D, Total_Bytes, hipMemcpyDeviceToHost);
	
	//double time_elapsed = getTime() - t0;

	double Gflops = (double)(niter * (1E-9 * n * n) * 28.0) / time_elapsed;
	double BW = (double)(niter * 1E-9 * (n * n * sizeof(double) * 4.0)) / time_elapsed;
	cout << "Number of Iterations        : " << niter << endl;
	cout << "Elapsed Time (sec)          : " << time_elapsed << endl;
	cout << "Sustained Gflops Rate       : " << Gflops << endl;
	cout << "Sustained Bandwidth (GB/sec): " << BW << endl<< endl;

	double mx;
	double l2norm = stats(E_prev, m, n, &mx);
	cout << "Max: " << mx << " L2norm: " << l2norm << endl;

	l2norm = stats1D(E_prev_1D, m, n, &mx, WIDTH);
	cout << "Max: " << mx << " L2norm: " << l2norm << " (1D or GPU)" <<endl;

	free(E);
	free(E_prev);
	free(R);

	hipHostFree(E_1D);
	hipHostFree(E_prev_1D);
	hipHostFree(R_1D);

	hipFree(d_E_1D);
	hipFree(d_E_prev_1D);
	hipFree(d_R_1D);

	return 0;
}

// ************************************************ Kernels Start ***************************************
__global__ void mirrorkernel(double *E_prev_1D, const int n, const int m, const int WIDTH){

	/* 
	* Copy data from boundary of the computational box 
	* to the padding region, set up for differencing
	* on the boundary of the computational box
	* Using mirror boundaries
	*/

	//int col, row;
	size_t row = blockIdx.y * blockDim.y + threadIdx.y + 1;
	size_t col = blockIdx.x * blockDim.x + threadIdx.x + 1;
	
	if (row <= m) {
		E_prev_1D[row * WIDTH + 0] = E_prev_1D[row * WIDTH + 2];
		E_prev_1D[row * WIDTH + (n + 1)] = E_prev_1D[row * WIDTH + (n - 1)];
	}
	
	if (col <= n) {

		E_prev_1D[0 * WIDTH + col] = E_prev_1D[2 * WIDTH + col];
		E_prev_1D[(m + 1) * WIDTH + col] = E_prev_1D[(m - 1) * WIDTH + col];
	}
}
__global__ void simulate_version1_PDE(const double alpha, const int n, const int m, const double dt,  double *E_1D, double *E_prev_1D, double *R_1D, const int WIDTH)
{

	int RADIUS = 1;
	int row = blockIdx.y * blockDim.y + threadIdx.y + RADIUS;
	int col = blockIdx.x * blockDim.x + threadIdx.x + RADIUS;

	if (row >= 1 && row <= m && col >= 1 && col <= n)
	{
		E_1D[row * WIDTH + col] = E_prev_1D[row * WIDTH + col] + alpha * (E_prev_1D[row * WIDTH + (col + 1)] + E_prev_1D[row * WIDTH + (col - 1)] - 4 * E_prev_1D[row * WIDTH + col] + E_prev_1D[(row + 1) * WIDTH + col] + E_prev_1D[(row - 1) * WIDTH + col]);
	}
}

__global__ void simulate_version1_ODE(const double alpha, const int n, const int m, const double dt,  double *E_1D, double *E_prev_1D, double *R_1D, const int WIDTH)
{
	int RADIUS = 1;
	int row = blockIdx.y * blockDim.y + threadIdx.y + RADIUS;
	int col = blockIdx.x * blockDim.x + threadIdx.x + RADIUS;
	int index = row * WIDTH + col;

	if (row >= 1 && row <= m && col >= 1 && col <= n)
	{
		E_1D[index] = E_1D[index] - dt * (kk * E_1D[index] * (E_1D[index] - a) * (E_1D[index] - 1) + E_1D[index] * R_1D[index]);
		R_1D[index] = R_1D[index] + dt * (epsilon + M1 * R_1D[index] / (E_1D[index] + M2)) * (-R_1D[index] - kk * E_1D[index] * (E_1D[index] - b - 1));
	}
}

// checkpoint 2
__global__ void simulate_version2(const double alpha, const int n, const int m, const double dt,  double *E_1D, double *E_prev_1D, double *R_1D, const int WIDTH)
{
	int RADIUS = 1;
	int row = blockIdx.y * blockDim.y + threadIdx.y + RADIUS;
	int col = blockIdx.x * blockDim.x + threadIdx.x + RADIUS;

	int index = row * WIDTH + col;

	if (row >= 1 && row <= m && col >= 1 && col <= n)
	{

		// PDE
		E_1D[row * WIDTH + col] = E_prev_1D[row * WIDTH + col] + alpha * (E_prev_1D[row * WIDTH + (col + 1)] + E_prev_1D[row * WIDTH + (col - 1)] - 4 * E_prev_1D[row * WIDTH + col] + E_prev_1D[(row + 1) * WIDTH + col] + E_prev_1D[(row - 1) * WIDTH + col]);

		//ODE
		E_1D[index] = E_1D[index] - dt * (kk * E_1D[index] * (E_1D[index] - a) * (E_1D[index] - 1) + E_1D[index] * R_1D[index]);
		R_1D[index] = R_1D[index] + dt * (epsilon + M1 * R_1D[index] / (E_1D[index] + M2)) * (-R_1D[index] - kk * E_1D[index] * (E_1D[index] - b - 1));
	}
}

// checkpoint 1
__global__ void simulate_version3(const double alpha, const int n, const int m, const double dt,  double *E_1D, double *E_prev_1D, double *R_1D, const int WIDTH)
{
	//int RADIUS = 1;
	int row = blockIdx.y * blockDim.y + threadIdx.y + 1;
	int col = blockIdx.x * blockDim.x + threadIdx.x + 1;
	int index = row * WIDTH + col;

	if (row >= 1 && row <= m && col >= 1 && col <= n)
	{
		double e_temp; //= E_1D[index];
		double r_temp = R_1D[index];
		double e_prev_temp = E_prev_1D[index];

		// PDE
		e_temp = e_prev_temp + alpha * (E_prev_1D[row * WIDTH + (col + 1)] + E_prev_1D[row * WIDTH + (col - 1)] - 4 * e_prev_temp + E_prev_1D[(row + 1) * WIDTH + col] + E_prev_1D[(row - 1) * WIDTH + col]);

		//ODE
		e_temp = e_temp - dt * (kk * e_temp * (e_temp - a) * (e_temp - 1) + e_temp * r_temp);
		r_temp = r_temp + dt * (epsilon + M1 * r_temp / (e_temp + M2)) * (-r_temp - kk * e_temp * (e_temp - b - 1));

		E_1D[index] = e_temp;
		R_1D[index] = r_temp;
	}
}


__global__ void simulate_version4(const double alpha, const int n, const int m, const double dt,  double *E_1D, double *E_prev_1D, double *R_1D, const int WIDTH)
{
	// __shared__ double tempR[(TILE_DIM + 2)*(TILE_DIM + 2)];
	__shared__ double tempE_prev[(TILE_DIM + 2)*(TILE_DIM + 2)];

	size_t LocalWidth = TILE_DIM + 2;

	// Global Indexing
	size_t row = blockIdx.y * blockDim.y + threadIdx.y + 1;
	size_t col = blockIdx.x * blockDim.x + threadIdx.x + 1;
	size_t index = row * WIDTH + col;

	size_t local_index = (threadIdx.y + 1)* LocalWidth + threadIdx.x + 1;

	// copy all
	if (row >= 1 && row <= m && col >= 1 && col <= n ){
		tempE_prev[local_index] = E_prev_1D[index];
	}
	
	// copy Right & Left 
	if (threadIdx.x + 1 == TILE_DIM){
		tempE_prev[local_index+1] = E_prev_1D[index+1];
		tempE_prev[local_index-TILE_DIM] = E_prev_1D[index-TILE_DIM];
	}

	// copy Up & Down
	if (threadIdx.y + 1== TILE_DIM){
		tempE_prev[local_index + LocalWidth] = E_prev_1D[index + WIDTH];
		tempE_prev[local_index - TILE_DIM*LocalWidth] = E_prev_1D[index - TILE_DIM*WIDTH];
	}
	
	// Make sure all threads get to this point before proceeding!
	__syncthreads(); // This will syncronize threads in a block

	if (row >= 1 && row <= m && col >= 1 && col <= n)
	{
		double e_temp;
		double r_temp = R_1D[index];
		
		// PDE
		e_temp = tempE_prev[local_index] + alpha * (tempE_prev[local_index + 1] + tempE_prev[local_index- 1] - 4 * tempE_prev[local_index] + tempE_prev[local_index + LocalWidth] + tempE_prev[local_index- LocalWidth]);

		//ODE
		e_temp = e_temp - dt * (kk * e_temp * (e_temp - a) * (e_temp - 1) + e_temp * r_temp);
		r_temp = r_temp + dt * (epsilon + M1 * r_temp / (e_temp + M2)) * (-r_temp - kk * e_temp * (e_temp - b - 1));
		
		E_1D[index] = e_temp;
		R_1D[index] = r_temp;
	}
}

void simV1(const double alpha, const int n, const int m,  const double dt,  int WIDTH, double* time, double *d_E_1D, double *d_E_prev_1D, double *d_R_1D)
{
	const dim3 block_size(TILE_DIM, TILE_DIM);
	const dim3 num_blocks(WIDTH / block_size.x, WIDTH / block_size.y);
	
	// Start the timer
	double t0 = getTime();
	simulate_version1_PDE<<<num_blocks, block_size>>>(alpha, n, m, dt, d_E_1D, d_E_prev_1D, d_R_1D, WIDTH);
	simulate_version1_ODE<<<num_blocks, block_size>>>(alpha, n, m, dt, d_E_1D, d_E_prev_1D, d_R_1D, WIDTH);
	hipStreamSynchronize(0);
	
	// end timer
	double time_elapsed = getTime() - t0;
	*time += time_elapsed;
}

void simV2(const double alpha, const int n, const int m,  const double dt,  int WIDTH, double* time, double *d_E_1D, double *d_E_prev_1D, double *d_R_1D)
{
	const dim3 block_size(TILE_DIM, TILE_DIM);
	const dim3 num_blocks(WIDTH / block_size.x, WIDTH / block_size.y);
	
	// Start the timer
	double t0 = getTime();

	simulate_version2<<<num_blocks, block_size>>>(alpha, n, m, dt, d_E_1D, d_E_prev_1D, d_R_1D, WIDTH);
	hipStreamSynchronize(0);

	double time_elapsed = getTime() - t0;
	*time += time_elapsed;
}

void simV3(const double alpha, const int n, const int m,  const double dt,  int WIDTH, double* time, double *d_E_1D, double *d_E_prev_1D, double *d_R_1D)
{
	const dim3 block_size(TILE_DIM, TILE_DIM);
	const dim3 num_blocks(WIDTH / block_size.x, WIDTH / block_size.y);
		
	// Start the timer
	double t0 = getTime();

	simulate_version3<<<num_blocks, block_size>>>(alpha, n, m, dt, d_E_1D, d_E_prev_1D, d_R_1D, WIDTH);
	hipStreamSynchronize(0);
	double time_elapsed = getTime() - t0;
	*time += time_elapsed;

}


void simV4(const double alpha, const int n, const int m,  const double dt,  int WIDTH, double* time, double *d_E_1D, double *d_E_prev_1D, double *d_R_1D)
{
	const dim3 block_size(TILE_DIM, TILE_DIM);
	const dim3 num_blocks(WIDTH / block_size.x, WIDTH / block_size.y);
	
	// Start the timer
	double t0 = getTime();
	
	simulate_version4<<<num_blocks, block_size>>>(alpha, n, m, dt, d_E_1D, d_E_prev_1D, d_R_1D, WIDTH);
	hipStreamSynchronize(0);
	double time_elapsed = getTime() - t0;
	*time += time_elapsed;
	
}

//************************************************* Kernels End *****************************************


// --------------------------------------------- Optimaztion Start-------------------------------------------------
__global__ void simulate_version5(const double alpha, const int n, const int m, const double dt,  double *E_1D, double *E_prev_1D, double *R_1D, const int WIDTH)
{
	int RADIUS = 1;
	int row = blockIdx.y * blockDim.y + threadIdx.y + RADIUS;
	int col = blockIdx.x * blockDim.x + threadIdx.x + RADIUS;

	int index = row * WIDTH + col;
	double t  = 0.0;
	int niter = 0;
	double T = 1000.0;

	while (t < T) {
		t += dt;
		niter++;

		if (row <= m) {
			E_prev_1D[row * WIDTH + 0] = E_prev_1D[row * WIDTH + 2];
			E_prev_1D[row * WIDTH + (n + 1)] = E_prev_1D[row * WIDTH + (n - 1)];
		}
		
		if (col <= n) {
	
			E_prev_1D[0 * WIDTH + col] = E_prev_1D[2 * WIDTH + col];
			E_prev_1D[(m + 1) * WIDTH + col] = E_prev_1D[(m - 1) * WIDTH + col];
		}
		
		__syncthreads();

		if (row >= 1 && row <= m && col >= 1 && col <= n)
		{

			// PDE
			E_1D[row * WIDTH + col] = E_prev_1D[row * WIDTH + col] + alpha * (E_prev_1D[row * WIDTH + (col + 1)] + E_prev_1D[row * WIDTH + (col - 1)] - 4 * E_prev_1D[row * WIDTH + col] + E_prev_1D[(row + 1) * WIDTH + col] + E_prev_1D[(row - 1) * WIDTH + col]);

			//ODE
			E_1D[index] = E_1D[index] - dt * (kk * E_1D[index] * (E_1D[index] - a) * (E_1D[index] - 1) + E_1D[index] * R_1D[index]);
			R_1D[index] = R_1D[index] + dt * (epsilon + M1 * R_1D[index] / (E_1D[index] + M2)) * (-R_1D[index] - kk * E_1D[index] * (E_1D[index] - b - 1));
			
			// double *tmp2 = E_1D;
			// E_1D = E_prev_1D;
			// E_prev_1D = tmp2;
			E_prev_1D[index] = E_1D[index];
		}

		//E_prev_1D[index] = E_1D[index];
		//if (row == 1 && col == 1) {
			//double *tmp2 = E_1D;
			//E_1D = E_prev_1D;
			//E_prev_1D = tmp2;
		//}
		__syncthreads();
	}

}

void simV5(const double alpha, const int n, const int m,  const double dt,  int WIDTH, double* time, double *d_E_1D, double *d_E_prev_1D, double *d_R_1D)
{
	const dim3 block_size(TILE_DIM, TILE_DIM);
	const dim3 num_blocks(WIDTH / block_size.x, WIDTH / block_size.y);
	
	// Start the timer
	double t0 = getTime();

	simulate_version5<<<num_blocks, block_size>>>(alpha, n, m, dt, d_E_1D, d_E_prev_1D, d_R_1D, WIDTH);
	hipStreamSynchronize(0);

	double time_elapsed = getTime() - t0;
	*time += time_elapsed;
}
// --------------------------------------------- Optimation End -------------------------------------------------

//================================================== Utilities =========================================

// Mirror Ghost Boundries
void mirrorBoundries(double *E_prev_1D, double* d_E_prev_1D, const int n, const int m, const int WIDTH){
	
	// ==================================================

	const dim3 block_size(TILE_DIM, TILE_DIM);
	const dim3 num_blocks(WIDTH / block_size.x, WIDTH / block_size.y);
	int Total_Bytes = WIDTH * WIDTH * sizeof(double);

	// Copy to GPU
	hipMemcpy(d_E_prev_1D, E_prev_1D, Total_Bytes, hipMemcpyHostToDevice);
		
	mirrorkernel<<<num_blocks, block_size>>>(d_E_prev_1D, n, m ,WIDTH);
				
	hipMemcpy(E_prev_1D, d_E_prev_1D, Total_Bytes, hipMemcpyDeviceToHost);
}

// Mirror Ghost Boundries
void mirrorBoundries(double *E_prev_1D, const int n, const int m, const int WIDTH)
{
	/* 
	* Copy data from boundary of the computational box 
	* to the padding region, set up for differencing
	* on the boundary of the computational box
	* Using mirror boundaries
	*/

	int col, row;
	for (row = 1; row <= m; row++)
	{
		//E_prev[row][0] = E_prev[row][2];
		E_prev_1D[row * WIDTH + 0] = E_prev_1D[row * WIDTH + 2];
	}

	for (row = 1; row <= m; row++)
	{
		//E_prev[row][n + 1] = E_prev[row][n - 1];
		E_prev_1D[row * WIDTH + (n + 1)] = E_prev_1D[row * WIDTH + (n - 1)];
	}

	for (col = 1; col <= n; col++)
	{
		//E_prev[0][col] = E_prev[2][col];
		E_prev_1D[0 * WIDTH + col] = E_prev_1D[2 * WIDTH + col];
	}

	for (col = 1; col <= n; col++)
	{
		//E_prev[m + 1][col] = E_prev[m - 1][col];
		E_prev_1D[(m + 1) * WIDTH + col] = E_prev_1D[(m - 1) * WIDTH + col];
	}
}

// Allocate a 2D array
double **alloc2D(int m, int n)
{
	double **E;
	int nx = n, ny = m;
	E = (double **)malloc(sizeof(double *) * ny + sizeof(double) * nx * ny);
	assert(E);
	int row;
	for (row = 0; row < ny; row++)
		E[row] = (double *)(E + ny) + row * nx;
	return (E);
}

/* Reports statistics about the computation
	These values should not vary (except to within roundoff)
 when we use different numbers of  processes to solve the problem
 */
double stats(double **E, int m, int n, double *_mx)
{
	double mx = -1;
	double l2norm = 0;
	int col, row;
	for (row = 1; row <= m; row++)
		for (col = 1; col <= n; col++)
		{
			l2norm += E[row][col] * E[row][col];
			if (E[row][col] > mx)
				mx = E[row][col];
		}
	*_mx = mx;
	l2norm /= (double)((m) * (n));
	l2norm = sqrt(l2norm);
	return l2norm;
}

double stats1D(double *E, int m, int n, double *_mx, int WIDTH)
{
	double mx = -1;
	double l2norm = 0;
	int col, row;
	int index = -1;
	for (row = 1; row <= m; row++)
	{
		for (col = 1; col <= n; col++)
		{
			index = row * WIDTH + col;
			l2norm += E[index] * E[index];
			if (E[index] > mx)
			{
				mx = E[index];
			}
		}
	}

	*_mx = mx;
	l2norm /= (double)((m) * (n));
	l2norm = sqrt(l2norm);
	return l2norm;
}

// Timer
// Make successive calls and take a difference to get the elapsed time.
static const double kMicro = 1.0e-6;
double getTime()
{
	struct timeval TV;
	struct timezone TZ;

	const int RC = gettimeofday(&TV, &TZ);
	if (RC == -1)
	{
		cerr << "ERROR: Bad call to gettimeofday" << endl;
		return (-1);
	}

	return (((double)TV.tv_sec) + kMicro * ((double)TV.tv_usec));
}

void cmdLine(int argc, char *argv[], double &T, int &n, int &px, int &py, int &plot_freq, int &no_comm, int &num_threads)
{
	/// Command line arguments
	// Default value of the domain sizes
	static struct option long_options[] = {
		{"n", required_argument, 0, 'n'},
		{"px", required_argument, 0, 'x'},
		{"py", required_argument, 0, 'y'},
		{"tfinal", required_argument, 0, 't'},
		{"plot", required_argument, 0, 'p'},
		{"nocomm", no_argument, 0, 'k'},
		{"numthreads", required_argument, 0, 'o'},
	};
	// Process command line arguments
	int ac;
	for (ac = 1; ac < argc; ac++)
	{
		int c;
		while ((c = getopt_long(argc, argv, "n:x:y:t:kp:o:", long_options, NULL)) != -1)
		{
			switch (c)
			{

				// Size of the computational box
			case 'n':
				n = atoi(optarg);
				break;

				// X processor geometry
			case 'x':
				px = atoi(optarg);

				// Y processor geometry
			case 'y':
				py = atoi(optarg);

				// Length of simulation, in simulated time units
			case 't':
				T = atof(optarg);
				break;
				// Turn off communication
			case 'k':
				no_comm = 1;
				break;

				// Plot the excitation variable
			case 'p':
				plot_freq = atoi(optarg);
				break;

				// Plot the excitation variable
			case 'o':
				num_threads = atoi(optarg);
				break;

				// Error
			default:
				printf("Usage: a.out [-n <domain size>] [-t <final time >]\n\t [-p <plot frequency>]\n\t[-px <x processor geometry> [-py <y proc. geometry] [-k turn off communication] [-o <Number of OpenMP threads>]\n");
				exit(-1);
			}
		}
	}
}