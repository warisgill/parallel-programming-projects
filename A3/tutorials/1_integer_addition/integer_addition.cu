
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add (int* a, int* b, int * c){

    *c = *a + *b;
    printf("\n value = %d \n", *c );
}

int main(){

    int a, b,c;
    int *d_a, *d_b, *d_c;
    int size = sizeof(int);

    a = 1;
    b = 1;

    // Allocate Space on Device (GPU)
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size); 

    // Copy Data to Device 
    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
    
    // Launch the kernel  on GPU
    add<<<4,4>>>(d_a,d_b,d_c);  // this kernel will be executed 16 times but the output value will remain the same (2)

    // Copy Results back to Host
    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

    


    printf("Result =  %d \n", c);

    // cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}

